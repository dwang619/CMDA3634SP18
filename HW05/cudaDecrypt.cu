#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__global__ void kernelDecrypt(unsigned n, unsigned int p, unsigned int g,
			      unsigned int h, unsigned int x) {

  int threadid = threadIdx.x;
  int blockid = blockIdx.x;
  int Nblock = blockDim.x;
  
  int id = threadid + blockid*Nblock;

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }
  }
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */ 
   
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
  
  //declare storage for an ElGamal cryptosytem
  unsigned int *h_x, *d_x;
  unsigned int n, p, g, h;
  unsigned int Nints;
  unsigned int N = atoi(argv[1]);

  h_x = (unsigned int *) malloc(N*sizeof(unsigned int));

  hipMalloc(&x, N*sizeof(unsigned int));
  
  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");
  
  
  cudaMemory(d_x,h_x,N*sizeof(unsigned int), hipMemcpyHostToDevice);
  
  // Read in the public key data from public_key.txt and the cyphertexts from messages.txt.
  FILE* publicKey = fopen("public_key.txt","r");
  fscanf(publicKey, "%u \n%u \n%u \n%u \n", &n, &p, &g, &h);
  fclose(publicKey);
  
  
  int Nthreads = atoi(argv[2]);
  int Nblocks= (N+Nthread-1)/Nthreads;
  
  double startTime = clock();
  
 
  kernelDecrypt <<<Nblocks , Nthreads>>>(n, p, g, h, x); 
  
  double endTime = clock();

  double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
  double work = (double) p;
  double throughput = work/totalTime;

  printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
 

  // After finding the secret key, decrypt the message
  FILE* msg = fopen("message.txt", "r");
  fscanf(msg, "%u\n", &Nints);
  unsigned int *m = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  for (int i = 0; i < Nints; i++) {
    fscanf(msg, "%u %u\n", &m[i], &a[i]);
  } 
  ElGamalDecrypt(m, a, Nints, p, x);

  unsigned char *message = (unsigned char *) malloc(1024*sizeof(unsigned char));
  unsigned int charsPerInt = sizeof(p) / sizeof(n);
  unsigned int Nchars = Nints * charsPerInt;
  convertZToString(m, Nints, message, Nchars);
  printf("The decrypted message is: %s\n", message);

  hipFree(d_x);
  free(h_x); 
  return 0;
}
